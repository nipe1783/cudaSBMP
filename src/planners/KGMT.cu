#include "hip/hip_runtime.h"
#include "planners/KGMT.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "agent/Agent.h"
#include "state/State.h"
#include "helper/helper.cuh"
#include "helper/helper.cu"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <ctime>
#include <hipcub/hipcub.hpp>
#include <filesystem>


#define SAMPLE_DIM 7
#define STATE_DIM 4
#define BLOCK_SIZE 32
#define NUM_R2 16
#define NUM_R1 16

KGMT::KGMT(float width, float height, int N, int n, int numIterations, int maxTreeSize, int numDisc, int sampleDim, float agentLength):
    width_(width), height_(height), N_(N), n_(n), numIterations_(numIterations), maxTreeSize_(maxTreeSize), numDisc_(numDisc), sampleDim_(sampleDim), agentLength_(agentLength){

    R1Size_ = width / N;
    R2Size_ = width / (n*N);

    d_G_ = thrust::device_vector<bool>(maxTreeSize);
    d_GNew_ = thrust::device_vector<bool>(maxTreeSize);
    d_U_ = thrust::device_vector<bool>(maxTreeSize);
    d_scanIdx_ = thrust::device_vector<int>(maxTreeSize);
    d_scanIdxGnew_= thrust::device_vector<int>(maxTreeSize);
    d_R1scanIdx_ = thrust::device_vector<int>(N*N);
    d_activeIdx_ = thrust::device_vector<int>(maxTreeSize);
    d_activeUIdx_ = thrust::device_vector<int>(maxTreeSize);
    d_activeR1Idx_ = thrust::device_vector<int>(N*N);
    d_treeParentIdx_ = thrust::device_vector<int>(maxTreeSize);
    d_treeSamples_ = thrust::device_vector<float>(maxTreeSize * sampleDim);
    d_xGoal_ = thrust::device_vector<float>(sampleDim);
    d_unexploredSamples_ = thrust::device_vector<float>(maxTreeSize * sampleDim);
    d_uParentIdx_ = thrust::device_vector<int>(maxTreeSize);
    d_R1Avail_ = thrust::device_vector<int>(N*N);
    d_R2Avail_ = thrust::device_vector<int>(N*N*n*n);
    d_R1Score_ = thrust::device_vector<float>(N*N);
    d_R1Valid_ = thrust::device_vector<int>(N*N);
    d_R2Valid_ = thrust::device_vector<int>(N*N*n*n);
    d_R1Invalid_ = thrust::device_vector<int>(N*N);
    d_R2Invalid_ = thrust::device_vector<int>(N*N*n*n);
    d_R1_ = thrust::device_vector<int>(N*N);
    d_R2_ = thrust::device_vector<int>(N*N*n*n);
    d_uValid_ = thrust::device_vector<bool>(maxTreeSize);

    d_G_ptr_ = thrust::raw_pointer_cast(d_G_.data());
    d_GNew_ptr_ = thrust::raw_pointer_cast(d_GNew_.data());
    d_treeSamples_ptr_ = thrust::raw_pointer_cast(d_treeSamples_.data());
    d_scanIdx_ptr_ = thrust::raw_pointer_cast(d_scanIdx_.data());
    d_scanIdxGnew_ptr_ = thrust::raw_pointer_cast(d_scanIdxGnew_.data());
    d_R1scanIdx_ptr_ = thrust::raw_pointer_cast(d_R1scanIdx_.data());
    d_activeIdx_ptr_ = thrust::raw_pointer_cast(d_activeIdx_.data());
    d_activeUIdx_ptr_ = thrust::raw_pointer_cast(d_activeUIdx_.data());
    d_activeR1Idx_ptr_ = thrust::raw_pointer_cast(d_activeR1Idx_.data());
    d_treeParentIdx_ptr_ = thrust::raw_pointer_cast(d_treeParentIdx_.data());
    d_xGoal_ptr_ = thrust::raw_pointer_cast(d_xGoal_.data());
    d_unexploredSamples_ptr_ = thrust::raw_pointer_cast(d_unexploredSamples_.data());
    d_uParentIdx_ptr_ = thrust::raw_pointer_cast(d_uParentIdx_.data());
    d_U_ptr_ = thrust::raw_pointer_cast(d_U_.data());
    d_R1Score_ptr_ = thrust::raw_pointer_cast(d_R1Score_.data());
    d_R1Avail_ptr_ = thrust::raw_pointer_cast(d_R1Avail_.data());
    d_R2Avail_ptr_ = thrust::raw_pointer_cast(d_R2Avail_.data());
    d_R1_ptr_ = thrust::raw_pointer_cast(d_R1_.data());
    d_R2_ptr_ = thrust::raw_pointer_cast(d_R2_.data());
    d_uValid_ptr_ = thrust::raw_pointer_cast(d_uValid_.data());
    d_R1Valid_ptr_ = thrust::raw_pointer_cast(d_R1Valid_.data());
    d_R2Valid_ptr_ = thrust::raw_pointer_cast(d_R2Valid_.data());
    d_R1Invalid_ptr_ = thrust::raw_pointer_cast(d_R1Invalid_.data());
    d_R2Invalid_ptr_ = thrust::raw_pointer_cast(d_R2Invalid_.data());


    hipMalloc(&d_costToGoal, sizeof(float));
    thrust::fill(d_treeParentIdx_.begin(), d_treeParentIdx_.end(), -1);
    thrust::fill(d_uParentIdx_.begin(), d_uParentIdx_.end(), -1);
    thrust::fill(d_R1Score_.begin(), d_R1Score_.end(), 1.0);

}

void KGMT::plan(float* initial, float* goal) {
    
    double t_kgmtStart = std::clock();
    
    // initialize vectors with root of tree
    hipMemcpy(d_treeSamples_ptr_, initial, sampleDim_ * sizeof(float), hipMemcpyHostToDevice);
    bool value = true;
    hipMemcpy(d_G_ptr_, &value, sizeof(bool), hipMemcpyHostToDevice);
    int r1_0 = getR1(initial[0], initial[1], R1Size_, N_);
    int r2_0 = getR2(initial[0], initial[1], r1_0, R1Size_, N_, R2Size_, n_);
    thrust::device_ptr<int> d_R1_ptr = d_R1_.data();
    thrust::device_ptr<int> d_R1Avail_ptr = d_R1Avail_.data();
    thrust::device_ptr<int> d_R2Avail_ptr = d_R2Avail_.data();
    thrust::device_ptr<int> d_R1Valid_ptr = d_R1Valid_.data();
    thrust::fill(d_R1_ptr + r1_0, d_R1_ptr + r1_0 + 1, 1);
    thrust::fill(d_R1Avail_ptr + r1_0, d_R1Avail_ptr + r1_0 + 1, 1);
    thrust::fill(d_R2Avail_ptr + r2_0, d_R2Avail_ptr + r2_0 + 1, 1);
    thrust::fill(d_R1Valid_ptr + r1_0, d_R1Valid_ptr + r1_0 + 1, 1);
    R1Threshold_ = 0.01;

    // initialize xGoal
    hipMemcpy(d_xGoal_ptr_, goal, sampleDim_ * sizeof(float), hipMemcpyHostToDevice);
    
    const int blockSize = 128;
	const int gridSize = std::min((maxTreeSize_ + blockSize - 1) / blockSize, 2147483647);
    int gridSizeActive = 1;
    int blockSizeActive = 32;

    // initialize random seed for hiprand
    hiprandState* d_randomStates;
    hipMalloc(&d_randomStates, maxTreeSize_ * sizeof(hiprandState));
    initCurandStates<<<(maxTreeSize_ + blockSize - 1) / blockSize, blockSize>>>(d_randomStates, maxTreeSize_, time(NULL));

    int itr = 0;
    treeSize_ = 1;
    int activeSize = 0;
    int maxIndex;
    float maxValue;
    while(itr < numIterations_){
        itr++;

        // UPDATE GRID SCORES:
        thrust::exclusive_scan(d_R1Avail_.begin(), d_R1Avail_.end(), d_R1scanIdx_.begin(), 0, thrust::plus<int>());
        activeSize = d_R1scanIdx_[N_*N_-1];
        (d_R1Avail_[N_*N_ - 1]) == 1 ? ++activeSize : 0;
        findInd<<<gridSize, blockSize>>>(
            N_*N_, 
            d_R1Avail_ptr_, 
            d_R1scanIdx_ptr_, 
            d_activeR1Idx_ptr_);
        updateR1<<<1, N_*N_>>>(
            d_R1Score_ptr_, 
            d_R1Avail_ptr_, 
            d_R2Avail_ptr_,
            d_R1Valid_ptr_,
            d_R1Invalid_ptr_,
            d_R1_ptr_,
            n_, 
            0.01, 
            R2Size_*R2Size_);

        // PROPAGATE G:
        thrust::exclusive_scan(d_G_.begin(), d_G_.end(), d_scanIdx_.begin(), 0, thrust::plus<int>());
        activeSize = d_scanIdx_[maxTreeSize_-1];
        (d_G_[maxTreeSize_ - 1]) ? ++activeSize : 0;
        
        findInd<<<gridSize, blockSize>>>(
            maxTreeSize_, 
            d_G_ptr_, 
            d_scanIdx_ptr_, 
            d_activeIdx_ptr_);

        
        blockSizeActive = 32;
        gridSizeActive = std::min(activeSize, int(floor(maxTreeSize_ / blockSizeActive)));
        propagateG<<<gridSizeActive, blockSizeActive>>>(
            activeSize, 
            d_activeIdx_ptr_, 
            d_G_ptr_,
            d_GNew_ptr_,
            d_treeSamples_ptr_, 
            d_unexploredSamples_ptr_,
            d_uParentIdx_ptr_,
            d_R1Valid_ptr_,
            d_R2Valid_ptr_,
            d_R1Invalid_ptr_,
            d_R2Invalid_ptr_,
            d_R1_ptr_,
            d_R2_ptr_,
            d_R1Avail_ptr_,
            d_R2Avail_ptr_,
            N_,
            n_,
            R1Size_,
            R2Size_,
            d_randomStates, 
            numDisc_, 
            agentLength_,
            R1Threshold_,
            d_R1Score_ptr_,
            itr);
        
        // UPDATE G:
        thrust::exclusive_scan(d_GNew_.begin(), d_GNew_.end(), d_scanIdx_.begin(), 0, thrust::plus<int>());
        activeSize = d_scanIdx_[maxTreeSize_-1];
        (d_GNew_[maxTreeSize_ - 1]) ? ++activeSize : 0;
        findInd<<<gridSize, blockSize>>>(
            maxTreeSize_, 
            d_GNew_ptr_, 
            d_scanIdx_ptr_, 
            d_activeIdx_ptr_);
        gridSizeActive = std::min(activeSize, int(floor(maxTreeSize_ / blockSizeActive)));
        blockSizeActive = 128;
        updateG<<<gridSizeActive, blockSizeActive>>>(
            d_treeSamples_ptr_, 
            d_unexploredSamples_ptr_, 
            d_uParentIdx_ptr_,
            d_treeParentIdx_ptr_,
            d_G_ptr_,
            d_GNew_ptr_,
            d_activeIdx_ptr_, 
            activeSize, 
            treeSize_);
        
        treeSize_ += activeSize;

        // printf("treeSize is %d\n", treeSize_);
        hipMemcpy(&costToGoal_, d_costToGoal, sizeof(float), hipMemcpyDeviceToHost);
        if(treeSize_ >= maxTreeSize_){
            // printf("Tree size exceeded maxTreeSize\n");
            break;
        }

        
        std::ostringstream filename;
        std::filesystem::create_directories("Data");
        std::filesystem::create_directories("Data/Samples");
        std::filesystem::create_directories("Data/Parents");
        std::filesystem::create_directories("Data/R1Scores");
        std::filesystem::create_directories("Data/R1Avail");
        std::filesystem::create_directories("Data/R1");
        filename.str("");
        filename << "Data/Samples/samples" << itr << ".csv";
        copyAndWriteVectorToCSV(d_treeSamples_, filename.str(), maxTreeSize_, sampleDim_);
        filename.str("");
        filename << "Data/Parents/parents" << itr << ".csv";
        copyAndWriteVectorToCSV(d_treeParentIdx_, filename.str(), maxTreeSize_, 1);
        filename.str("");
        filename << "Data/R1Scores/R1Scores" << itr << ".csv";
        copyAndWriteVectorToCSV(d_R1Score_, filename.str(), N_*N_, 1);
        filename.str("");
        filename << "Data/R1Avail/R1Avail" << itr << ".csv";
        copyAndWriteVectorToCSV(d_R1Avail_, filename.str(), N_*N_, 1);
        filename.str("");
        filename << "Data/R1/R1" << itr << ".csv";
        copyAndWriteVectorToCSV(d_R1_, filename.str(), N_*N_, 1);
    }

    double t_kgmt = (std::clock() - t_kgmtStart) / (double) CLOCKS_PER_SEC;
    std::cout << "time inside KGMT is " << t_kgmt << std::endl;

    // move vectors to csv to be plotted.
    copyAndWriteVectorToCSV(d_treeSamples_, "samples.csv", maxTreeSize_, sampleDim_);
    copyAndWriteVectorToCSV(d_unexploredSamples_, "unexploredSamples.csv", maxTreeSize_, sampleDim_);
    copyAndWriteVectorToCSV(d_treeParentIdx_, "parentRelations.csv", maxTreeSize_, 1);
    copyAndWriteVectorToCSV(d_uParentIdx_, "uParentIdx.csv", maxTreeSize_, 1);
    copyAndWriteVectorToCSV(d_G_, "G.csv", maxTreeSize_, 1);
    copyAndWriteVectorToCSV(d_R2Avail_, "R2Avail.csv", N_*N_*n_*n_, 1);
    copyAndWriteVectorToCSV(d_R1Avail_, "R1Avail.csv", N_*N_, 1);
    copyAndWriteVectorToCSV(d_R1Valid_, "R1Valid.csv", N_*N_, 1);
    copyAndWriteVectorToCSV(d_R2Valid_, "R2Valid.csv", N_*N_*n_*n_, 1);
    copyAndWriteVectorToCSV(d_R1Invalid_, "R1Invalid.csv", N_*N_, 1);
    copyAndWriteVectorToCSV(d_R2Invalid_, "R2Invalid.csv", N_*N_*n_*n_, 1);
    copyAndWriteVectorToCSV(d_R1Score_, "R1Score.csv", N_*N_, 1);
    copyAndWriteVectorToCSV(d_R1_, "R1.csv", N_*N_, 1);

    // Free the allocated memory for hiprand states
    hipFree(d_randomStates);
}

__global__
void findInd(int numSamples, bool* S, int* scanIdx, int* activeS){
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= numSamples)
        return;
    if (!S[node]) {
        return;
    }
    activeS[scanIdx[node]] = node;
}

__global__
void findInd(int numSamples, int* S, int* scanIdx, int* activeS){
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= numSamples)
        return;
    if (!S[node]) {
        return;
    }
    activeS[scanIdx[node]] = node;
}

__global__ void propagateG(
    int sizeG, 
    int* activeGIdx, 
    bool* G,
    bool* GNew,
    float* treeSamples,
    float* unexploredSamples,
    int* uParentIdx,
    int* R1Valid,
    int* R2Valid,
    int* R1Invalid,
    int* R2Invalid,
    int* R1,
    int* R2,
    int* R1Avail,
    int* R2Avail,
    int N,
    int n,
    float R1Size,
    float R2Size,
    hiprandState* randomStates,
    int numDisc,
    float agentLength,
    float R1Threshold,
    float* R1Scores,
    int itr) {

    // block expands x0 BLOCK_SIZE times.
    if (blockIdx.x >= sizeG)
        return;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int x0Idx;
    if(threadIdx.x == 0){
        x0Idx = activeGIdx[blockIdx.x];
        G[x0Idx] = false;
    }
    __syncthreads();
    __shared__ float x0[SAMPLE_DIM];
    if(threadIdx.x < SAMPLE_DIM){
        x0[threadIdx.x] = treeSamples[x0Idx * SAMPLE_DIM + threadIdx.x];
    }
    __syncthreads();

    hiprandState randState = randomStates[tid];
    float* x1 = &unexploredSamples[tid * SAMPLE_DIM];
    uParentIdx[tid] = x0Idx;
    bool valid = propagateAndCheck(x0, x1, numDisc, agentLength, &randState);
    int r1 = getR1(x1[0], x1[1], R1Size, N);
    int r2 = getR2(x1[0], x1[1], r1, R1Size, N , R2Size, n);
    atomicAdd(&R1[r1], 1);
    atomicAdd(&R2[r2], 1);
    if(valid){
        if(R1Scores[r1] > R1Threshold){
            GNew[tid] = true;
        }
        if(R1Avail[r1] == 0){
            atomicExch(&R1Avail[r1], 1);
        }
        if(R2Avail[r2] == 0){
            atomicExch(&R2Avail[r2], 1);
        }
        atomicAdd(&R2Valid[r2], 1);
        atomicAdd(&R1Valid[r1], 1);
    }
    else {
        atomicAdd(&R1Invalid[r1], 1);
        atomicAdd(&R2Invalid[r2], 1);
    }
    randomStates[tid] = randState;

}

// 1 Block Version. Each thread calculates 1 R1 cell.
// TODO: Change it to a 2D block. each thread square calculates 1 R1 cell. Should help with fetching R2Avail.
__global__ void updateR1(
    float* R1Score, 
    int* R1Avail, 
    int* R2Avail, 
    int* R1Valid, 
    int* R1Invalid,
    int* R1,
    int n, 
    float epsilon, 
    float R1Vol) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NUM_R1 * NUM_R1)
        return;

    // Use shared memory for reduction
    __shared__ float s_totalSum;

    float score = 0.0f;
    if (R1Avail[tid] != 0) {
        int nValid = R1Valid[tid];
        float covR = 0;
        for (int i = tid * n * n; i < (tid + 1) * n * n; i++) {
            covR += R2Avail[i];
        }
        covR /= n * n;

        float freeVol = ((epsilon + nValid) / (epsilon + nValid + R1Invalid[tid]));
        score = pow(freeVol, 4) / ((1 + covR) * (1 + pow(R1[tid], 2)));
    }

    typedef hipcub::BlockReduce<float, NUM_R1*NUM_R1> BlockReduceFloatT;
    __shared__ typename BlockReduceFloatT::TempStorage tempStorageFloat;
    float blockSum = BlockReduceFloatT(tempStorageFloat).Sum(score);

    if (threadIdx.x == 0) {
        s_totalSum = blockSum;
    }
    __syncthreads();

    // Normalize the score
    if(R1Avail[tid] == 0){
        R1Score[tid] = 1.0f;
    }
    else {
        R1Score[tid] = score / s_totalSum;
    }
}

__global__ void updateG(
    float* treeSamples, 
    float* unexploredSamples, 
    int* unexploredParentIdx,
    int* treeParentIdx,
    bool* G,
    bool* GNew,
    int* GNewIdx, 
    int GNewSize, 
    int treeSize){
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= GNewSize)
        return;

    // move valid unexplored sample to tree:
    int x1TreeIdx = treeSize + tid;
    int x1UnexploredIdx = GNewIdx[tid];
    treeParentIdx[x1TreeIdx] = unexploredParentIdx[x1UnexploredIdx];
    treeSamples[x1TreeIdx * SAMPLE_DIM] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 1] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 1];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 2] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 2];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 3] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 3];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 4] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 4];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 5] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 5];
    treeSamples[x1TreeIdx * SAMPLE_DIM + 6] = unexploredSamples[x1UnexploredIdx * SAMPLE_DIM + 6];

    // update G:
    G[x1TreeIdx] = true;
    GNew[tid] = false;
}

__global__ void initCurandStates(hiprandState* states, int numStates, int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numStates)
        return;
    hiprand_init(seed, tid, 0, &states[tid]);
}

__device__
bool propagateAndCheck(float* x0, float* x1, int numDisc, float agentLength, hiprandState* state){
    // Generate random controls
    float a = hiprand_uniform(state) * 20.0f - 10.0f;  // a between -5 and 5
    float steering = hiprand_uniform(state) * 2.0f * M_PI - M_PI;  // steering between -π and π
    float duration = hiprand_uniform(state) * .4f + 0.1f;  // duration between 0.1 and 0.5

    float dt = duration / numDisc;
    float x = x0[0];
    float y = x0[1];
    float theta = x0[2];
    float v = x0[3];

    float cos_theta, sin_theta, tan_steering;

    for (int i = 0; i < numDisc; i++) {
        cos_theta = cosf(theta);
        sin_theta = sinf(theta);
        tan_steering = tanf(steering);

        x += v * cos_theta * dt;
        y += v * sin_theta * dt;
        theta += (v / agentLength) * tan_steering * dt;
        v += a * dt;
    }

    x1[0] = x;
    x1[1] = y;
    x1[2] = theta;
    x1[3] = v;
    x1[4] = a;
    x1[5] = steering;
    x1[6] = duration;
    //TODO: Update this.
    return true;
}

__host__ __device__ int getR1(float x, float y, float R1Size, int N) {
    int cellX = static_cast<int>(x / R1Size);
    int cellY = static_cast<int>(y / R1Size);
    if (cellX >= 0 && cellX < N && cellY >= 0 && cellY < N) {
        return cellY * N + cellX;
    }
    return -1; // Invalid cell
}
__host__ __device__ int getR2(float x, float y, int r1, float R1Size, int N, float R2Size, int n) {
    if (r1 == -1) {
        return -1; // Invalid R1 cell, so R2 is also invalid
    }

    int cellY_R1 = r1 / N;
    int cellX_R1 = r1 % N;

    // Calculate the local coordinates within the R1 cell
    float localX = x - cellX_R1 * R1Size;
    float localY = y - cellY_R1 * R1Size;

    int cellX_R2 = static_cast<int>(localX / R2Size);
    int cellY_R2 = static_cast<int>(localY / R2Size);
    if (cellX_R2 >= 0 && cellX_R2 < n && cellY_R2 >= 0 && cellY_R2 < n) {
        int localR2 = cellY_R2 * n + cellX_R2;
        return r1 * (n * n) + localR2; // Flattened index
    }
    return -1; // Invalid subcell
}
