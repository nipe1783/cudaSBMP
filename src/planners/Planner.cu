#include "planners/Planner.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "agent/Agent.h"
#include "state/State.h"
#include <hiprand/hiprand_kernel.h>
#include <chrono>


void Planner::checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

