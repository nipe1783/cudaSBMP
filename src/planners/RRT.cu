#include "planners/RRT.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <Eigen/Core>
#include "agent/Agent.h"

void RRT::plan(float* start, float* goal){
    
    float* samples = nullptr;
    float* controls = nullptr;
    generateRandomTree(start, 100, &samples, &controls);
}