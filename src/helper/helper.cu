#include "helper/helper.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

// Templated function to print contents of a thrust device vector given a raw pointer and size
template <typename T>
void printDeviceVector(const T* d_ptr, int size) {
    thrust::host_vector<T> h_vec(size);
    hipMemcpy(thrust::raw_pointer_cast(h_vec.data()), d_ptr, size * sizeof(T), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; ++i) {
        std::cout << h_vec[i] << " ";
    }
    std::cout << std::endl;
}

__device__ void printSample(float* x, int sampleDim) {
    for (int i = 0; i < sampleDim; ++i) {
        printf("%f ", x[i]);
    }
    printf("\n");
}

template <typename T>
void writeVectorToCSV(const thrust::host_vector<T>& vec, const std::string& filename, int rows, int cols) {
    std::ofstream file;
    file.open(filename);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            file << vec[i * cols + j];
            if (j < cols - 1) {
                file << ",";
            }
        }
        file << std::endl;
    }

    file.close();
}

template <typename T>
void copyAndWriteVectorToCSV(const thrust::device_vector<T>& d_vec, const std::string& filename, int rows, int cols) {
    thrust::host_vector<T> h_vec(d_vec.size());
    hipMemcpy(thrust::raw_pointer_cast(h_vec.data()), thrust::raw_pointer_cast(d_vec.data()), d_vec.size() * sizeof(T), hipMemcpyDeviceToHost);
    writeVectorToCSV(h_vec, filename, rows, cols);
}